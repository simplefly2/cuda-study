﻿#include <stdio.h>
#include <stdlib.h>

#include <cmath>
#include <cfloat>
#include "hip/hip_runtime.h"

#define CHECK(func) do \
{ \
	hipError_t err = func \
	if (err != hipSuccess) \
	{   \
		printf("error: %s, %d \n", hipGetErrorString(err), __LINE__); \
	} while(0) \
}

#define BLOCK_SIZE 256
#define WARP_SIZE 32

#define M 1024
#define N 2048


/*
Softmax:
	
	主要是两步 reduce:

	1. 求每行最大值 max_val:

	2. 求每行指数和：
	
		sum = sum(expf(xi - max_val))

	3. 求值：
		
		xi = exp(xi-max_val) / sum
*/

/*
	input, output shape: (M, N)
*/

void cpu_softmax(float* input, float* output, int m, int n)
{
	
	for (int i = 0; i < m; ++i)
	{
		// max_val:
		float max_val = 0.0f;

		for (int j = 0; j < n; ++j)
		{
			max_val = fmaxf(input[i * n + j], max_val);
		}

		// sum:
		float sum = 0.0f;
		for (int j = 0; j < n; ++j)
		{
			sum += expf(input[i * n + j] - max_val);
		}

		// value:
		for (int j = 0; j < n; ++j)
		{
			output[i*n+j] = expf(input[i * n + j] - max_val) / sum;
		}
	}

}


/////////////////////////  kernel0
//  每行并行计算， 一个线程处理一行数据：
__global__ void softmaxThread(float* input, float* output, int m, int n)
{
	// int index = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.x*blockDim.x + threadIdx.x;

	if (row < m)
	{
		float* inp = input + row * n;

		// max_val:
		float max_val = 0.0f;

		for (int j = 0; j < n; ++j)
		{
			max_val = fmaxf(inp[j], max_val);
		}

		// sum:
		float sum = 0.0f;
		for (int j = 0; j < n; ++j)
		{
			sum += expf(inp[j] - max_val);
		}

		// value:
		for (int j = 0; j < n; ++j)
		{
			output[row*n + j] = expf(inp[j] - max_val) / sum;
		}
	}

}


////////////////////////////////// kernel1: 
// 一个block处理一行数据：
// block_reduce (normal reduce)
__global__ void softmaxBlockNormal(float* input, float* output, int m, int n)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;

	__shared__ float sdata[BLOCK_SIZE];

	int row = bid;
	if (row < m)
	{
		float* inp = input + row * n;

		// max_val:
		float max_val = -FLT_MAX;
		for (int i = tid; i < n; i += blockDim.x)
		{
			max_val = fmaxf(max_val, inp[i]);
		}

		sdata[tid] = max_val;

		__syncthreads();

		for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
		{
			if (tid < stride)
				sdata[tid] = fmaxf(sdata[tid], sdata[tid + stride]);

			__syncthreads();
		}

		max_val = sdata[0];

		// sum:
		float sum = 0.0f;
		for (int i = tid; i < n; i += blockDim.x)
		{
			sum += expf(inp[i]-max_val);
		}

		sdata[tid] = sum;
		__syncthreads();

		for (int stride = blockDim.x / 2; stride > 0; stride /= 2)
		{
			if (tid < stride)
				sdata[tid] = sdata[tid] + sdata[tid + stride];

			__syncthreads();
		}

		sum = sdata[0];

		// value:
		for (int i = tid; i < n; i += blockDim.x)
		{
			output[row*n+i] = expf(inp[i] - max_val) / sum;
		}

	}

}


////////////////////////// kernel2: 
// 一个block处理一行数据：
// block_reduce (based warp reduce)
__device__ float warpMax(float val)
{
	for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1)
	{
		val = fmaxf(val, __shfl_down_sync(0xffffffff, val, offset));
	}

	return __shfl_sync(0xffffffff, val, 0);
}

__device__ float warpSum(float sum)
{
	for (int offset = WARP_SIZE / 2; offset > 0; offset >>= 1)
	{
		sum += __shfl_down_sync(0xffffffff, sum, offset);
	}

	return __shfl_sync(0xffffffff, sum, 0);
}


__global__ void softmaxBlockWarp(float* input, float* output, int m, int n)
{
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	int warpid = threadIdx.x / 32;
	int laneid = threadIdx.x % 32;

	__shared__ float warp_result[32];
	__shared__ float b_max;
	__shared__ float b_sum;

	int row = bid;
	if (row < m)
	{
		float* inp = input + row * n;
		
		// max_val:
		float max_val = -FLT_MAX;
		for (int i = tid; i < n; i += blockDim.x)
		{
			max_val = fmaxf(max_val, inp[i]);
		}

		max_val = warpMax(max_val);

		if (laneid == 0) warp_result[warpid] = max_val;
		__syncthreads();

		max_val = tid < 32 ? warp_result[tid] : -FLT_MAX;

		max_val = warpMax(max_val);

		if (tid == 0) b_max = max_val;
		__syncthreads();

		// sum:
		float sum = 0.0f;
		for (int i = tid; i < n; i += blockDim.x)
		{
			sum += expf(inp[i] - b_max);
		}

		sum = warpSum(sum);
		if (laneid == 0) warp_result[warpid] = sum;
		__syncthreads();

		sum = tid < 32 ? warp_result[tid] : 0.0f;
		sum = warpSum(sum);

		if (tid == 0) b_sum = sum;
		__syncthreads();

		// value:
		for (int i = tid; i < n; i += blockDim.x)
		{
			output[row*n+i] = expf(inp[i] - b_max) / b_sum;
		}
	}

}





int main()
{
	// cpu:
	float* input, * output, * gpu_output;

	size_t input_bytes = sizeof(float) * M * N;
	size_t output_bytes = sizeof(float) * M * N;

	input = (float*)malloc(input_bytes);
	output = (float*)malloc(output_bytes);
	gpu_output = (float*)malloc(output_bytes);

	for (unsigned i = 0; i < M; ++i)
	{
		for(unsigned j=0; j<N; ++j)
			input[i*N+j] = static_cast<float>(rand()) / RAND_MAX;
	}

	cpu_softmax(input, output, M, N);
	//float cpu_sum = 0.0;
	//for (unsigned i = 0; i < N; ++i)
	//{
	//	if(i<50)
	//		printf("i: %d, input: %.8f, output: %.8f \n", i, input[i], output[i]);
	//	cpu_sum += output[i];
	//}
	//printf("cpu sum: %.2f \n", cpu_sum);


	// gpu:
	float* d_input, *d_output;

	hipMalloc((void**)&d_input, input_bytes);
	hipMalloc((void**)&d_output, output_bytes);

	hipMemcpy(d_input, input, input_bytes, hipMemcpyHostToDevice);

	//unsigned grid_x = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
	//dim3 grid_size(grid_x);
	//dim3 block_size(BLOCK_SIZE);
	//softmaxThread<<<grid_size, block_size>>>(d_input, d_output, M, N);

	//unsigned grid_x = M;
	//dim3 grid_size(grid_x);
	//dim3 block_size(BLOCK_SIZE);
	//softmaxBlockNormal<<<grid_size, block_size >>>(d_input, d_output, M, N);

	unsigned grid_x = M;
	dim3 grid_size(grid_x);
	dim3 block_size(BLOCK_SIZE);
	softmaxBlockWarp<<<grid_size, block_size >>>(d_input, d_output, M, N);


	hipMemcpy(gpu_output, d_output, output_bytes, hipMemcpyDeviceToHost);

	//// check:
	bool error = false;
	for (unsigned i = 0; i < M; ++i)
	{
		for (unsigned j = 0; j < N; ++j)
		{
			if (fabs(gpu_output[i*N+j] - output[i*N+j]) > 1e-8)
			{
				error = true;
			}

		}

	}

	printf("result: %s \n", error ? "fail" : "pass");

	int offset = 20;
	for (unsigned i = 0; i < offset + 20; ++i)
	{
		printf("gpu: %.8f, cpu: %.8f \n", gpu_output[i], output[i]);
	}

	// free:
	free(input);
	free(output);
	free(gpu_output);

	hipFree(d_input);
	hipFree(d_output);

	return 0;


}