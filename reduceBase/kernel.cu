﻿#include "stdio.h"
#include "stdlib.h"
#include <cmath>

#include "hip/hip_runtime.h"


#define M 1024
#define N 768

#define WARP_SIZE 32
#define BLOCK_SIZE 256


/*
	
	计算矩阵 M* N 每行之和

*/ 

template <typename T>
void cpu_MatrixRowSum(T* input, T* output, int m, int n)
{
	for (int i = 0; i < m; ++i)
	{
		T sum = 0;
		for (int j = 0; j < n; ++j)
			sum += input[i * n + j];
		
		output[i] = sum;
	}
}



template <typename T>
__device__ T warpReduce(T val)
{
	for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2)
		val += __shfl_down_sync(0xffffffff, val, offset);

	return val;
}


// 每个warp归约一行数据：
template <typename T>
__global__ void matrixRowSum(T* input, T* output, int m, int n)
{
	int warpid = threadIdx.x / WARP_SIZE;
	int laneid = threadIdx.x % WARP_SIZE;
	int warp_num = blockDim.x / WARP_SIZE;

	int row = blockIdx.x * warp_num + warpid;
	if (row < m)
	{
		T* inp = input + blockIdx.x * warp_num * n;

		T sum = (T)0.0f;
		for (int i = laneid; i < n; i += WARP_SIZE)
		{
			sum += inp[i];
		}

		sum = warpReduce<T>(sum);


		if (laneid == 0)
			output[row] = sum;
	}

}


int main()
{
	// cpu:
	int* input, * output, *gpu_output;
	size_t input_bytes = sizeof(int) * M * N;
	size_t output_bytes = sizeof(int) * M;

	input = (int*)malloc(input_bytes);
	output = (int*)malloc(output_bytes);
	gpu_output = (int*)malloc(output_bytes);

	for (unsigned int i = 0; i < M; ++i)
	{
		for (unsigned int j = 0; j < N; ++j)
		{
			input[i * N + j] = rand() % 1024;
		}
	}

	cpu_MatrixRowSum(input, output, M, N);

	// gpu:
	int* d_input, * d_output;
	hipMalloc((void**)&d_input, input_bytes);
	hipMalloc((void**)&d_output, output_bytes);

	hipMemcpy(d_input, input, input_bytes, hipMemcpyHostToDevice);

	int grid_x = (M + BLOCK_SIZE - 1) / BLOCK_SIZE;
	printf("1111111111grid_x: %d \n", grid_x);
	dim3 grid_size(grid_x);
	dim3 block_size(BLOCK_SIZE);

	matrixRowSum<<<grid_size, block_size>>>(d_input, d_output, M, N);

	hipMemcpy(gpu_output, d_output, output_bytes, hipMemcpyDeviceToHost);


	// check:
	bool error = false;
	for (int i = 0; i < M; ++i)
	{
		if (fabs(output[i] - gpu_output[i]) > 1e-6)

			error = true;
	}

	printf("result: %s \n", error ? "fail" : "pass");

	int offset = 20;
	for (int i = offset; i < offset+20; ++i)
		printf("gpu: %d, cpu: %d \n", gpu_output[i], output[i]);


	// free
	free(input);
	free(output);
	free(gpu_output);
	hipFree(d_input);
	hipFree(d_output);


	return 0;

}